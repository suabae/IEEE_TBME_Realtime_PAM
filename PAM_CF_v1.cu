// written by Sua Bae (3/09/2021)

#include <hip/hip_runtime.h>


__constant__ float 			trans_aElePos[256]; // transducer element position
__constant__ unsigned int 		trans_nNumEle; // num of elements

__constant__ unsigned int 		rf_nSample; 			// num of RF samples
__constant__ unsigned int 		rf_nChannel;			// num of channels of RF data
__constant__ float 			rf_nOffsetDelay_m; 	// [m] offset of input data (to be added to tx delay)
__constant__ float 			rf_nMeter2Pixel;  	// [pixel/m]  = sampling frequency / nSoundSpeed; 

__constant__ unsigned int 		g_nXdim; 			// num of x points 
__constant__ unsigned int 		g_nZdim; 			// num of z points 
__constant__ float 			g_dx; 				// [m] pixel size in x
__constant__ float 			g_dz; 				// [m] pixel size in z
__constant__ float 			g_nXstart; 			// [m] x coordinate of the first lateral pixel
__constant__ float 			g_nZstart; 			// [m] z coordinate of the first axial pixel
__constant__ unsigned int 		g_nTdim; 			// num of time points to be integrated


//***	THREADBLOCK DIMENSION ***
//	ThreadBlockSize =[N,1,1];
//	GridSize = [ceil(g_nTdim/N),g_nXdim,g_nZdim];
__global__ void _PAM_CF(float* CavMap_tzx, float* RfData_sc)		
{
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int xidx = blockIdx.y;
	int zidx = blockIdx.z;
	
	if (tidx < g_nTdim) {	
        float nX = (float)xidx * g_dx + g_nXstart; // x position of image grid [m]
        float nZ = (float)zidx * g_dz + g_nZstart + 1e-20; // z position of image grid [m]				

        float nTxDelay_m; // [m] distance from FUS transducer to the imaging point (RX delay)
        float nRxDelay_m; // [m] distance from imaging point to element (RX delay)
        float nDelay_px; // [pixel] round trip delay 
        float nDelay_int; // [pixel] integer part of round trip delay 
        float nDelay_frc; // [pixel] fractional part of round trip delay 
        int nAdd; // address of the rf datum
        float nInterpVal; // interpolated value
        float nCompenVal; // spherical spreading compensated value
        float nCF; // coherence factor

        nTxDelay_m = nZ;			

        float nChannelSum = 0; // refresh
        float nSumOfSquared = 0;
        #pragma unroll	
        for (int cidx = 0; cidx < rf_nChannel; cidx++) { // channel index
            nRxDelay_m = sqrt((nX-trans_aElePos[cidx])*(nX-trans_aElePos[cidx]) + nZ*nZ); // [m] distance from imaging point to element
            nDelay_px = (nTxDelay_m + nRxDelay_m + rf_nOffsetDelay_m)*rf_nMeter2Pixel; // [sample]						

            nDelay_int = (int)(nDelay_px);
            nDelay_frc = nDelay_px - nDelay_int;
            nAdd = cidx*rf_nSample + nDelay_int + tidx; 
            nInterpVal =  RfData_sc[nAdd]*(1-nDelay_frc) + RfData_sc[nAdd+1]*nDelay_frc; // interpolated sample for each channel and time point
            nCompenVal = sqrt(nRxDelay_m)*nInterpVal;
            nChannelSum = nChannelSum + nCompenVal; // stack onto the channel sum
            nSumOfSquared = nSumOfSquared + nCompenVal*nCompenVal;
        }

        nCF = (nChannelSum*nChannelSum)/(rf_nChannel*nSumOfSquared);
        CavMap_tzx[tidx + zidx*g_nTdim + xidx*g_nTdim*g_nZdim] = (nCF*nChannelSum)*(nCF*nChannelSum);

	}
}

